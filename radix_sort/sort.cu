#include "hip/hip_runtime.h"
#include "sort.h"

#define MAX_BLOCK_SZ 1024
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \
	((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif

__global__
void gpu_build_pred(unsigned int* const d_out,
	unsigned int* const d_in,
	const size_t numElems,
	unsigned int bit_mask,
	unsigned int zero_or_one)
{
	unsigned int glbl_t_idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (glbl_t_idx >= numElems)
		return;

	unsigned int curr_elem = d_in[glbl_t_idx];
	// predicate is true if result is 0
	unsigned int pred = curr_elem & bit_mask;
	unsigned int pred_result = zero_or_one ? 0 : 1;
	if (pred == bit_mask)
	{
		pred_result = zero_or_one ? 1 : 0;
	}
	d_out[glbl_t_idx] = pred_result;

	__syncthreads();

	unsigned int dummy = d_out[glbl_t_idx];
}

__global__
void gpu_scatter_elems(unsigned int* const d_out,
	unsigned int* const d_in,
	unsigned int* const d_preds,
	unsigned int* const d_scanned_preds,
	unsigned int* const d_out_offset,
	const size_t numElems,
	unsigned int zero_or_one)
{
	unsigned int glbl_t_idx = blockDim.x * blockIdx.x + threadIdx.x;

	if (glbl_t_idx >= numElems || d_preds[glbl_t_idx] == 0)
	{
		return;
	}

	unsigned int d_out_idx = d_scanned_preds[glbl_t_idx];
	// offset the addresses with total sum of predicate 
	//  array when working with 1 bits
	if (zero_or_one == 1)
		d_out_idx = d_out_idx + *d_out_offset;
	unsigned int curr_val = d_in[glbl_t_idx];
	d_out[d_out_idx] = curr_val;
}

void radix_sort(unsigned int* const d_out,
	unsigned int* const d_in,
	unsigned int* const d_preds,
	unsigned int* const d_scanned_preds,
	const size_t numElems)
{
	unsigned int block_sz = 1024;
	// Instead of using ceiling and risking miscalculation due to precision, just automatically  
	//  add 1 to the grid size when the input size cannot be divided cleanly by the block's capacity
	//unsigned int grid_sz = (unsigned int)std::ceil((double)numElems / (double)block_sz);
	unsigned int grid_sz = numElems / block_sz;
	if (numElems % block_sz != 0)
		grid_sz += 1;

	unsigned int* d_scatter_offset;
	checkCudaErrors(hipMalloc(&d_scatter_offset, sizeof(unsigned int)));

	// Do this for every bit, from LSB to MSB
	for (unsigned int sw = 0; sw < (sizeof(unsigned int) * 8); ++sw)
	{
		for (unsigned int bit = 0; bit <= 1; ++bit)
		{
			unsigned int bit_mask = 1 << sw;

			// Build predicate array
			gpu_build_pred << <grid_sz, block_sz >> >(d_preds, d_in, numElems, bit_mask, bit);

			// Scan predicate array
			//  If working with 0's, make sure the total sum of the predicate 
			//  array is recorded for determining the offset of the 1's
			if (bit == 0)
				sum_scan_blelloch(d_scanned_preds, d_scatter_offset, d_preds, numElems);
			else
				sum_scan_blelloch(d_scanned_preds, NULL, d_preds, numElems);

			// Scatter d_in's elements to their new locations in d_out
			//  Use predicate array to figure out which threads will move
			//  Use scanned predicate array to figure out the locations
			gpu_scatter_elems << <grid_sz, block_sz >> >(d_out, d_in, d_preds, d_scanned_preds, d_scatter_offset, numElems, bit);
		}

		// Copy d_out to d_in in preparation for next significant bit
		checkCudaErrors(hipMemcpy(d_in, d_out, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
	}

	checkCudaErrors(hipFree(d_scatter_offset));
}

__global__ void gpu_radix_sort_local(unsigned int* d_out_sorted,
	unsigned int* d_prefix_sums,
	unsigned int* d_block_sums,
	unsigned int input_shift_width,
	unsigned int* d_in,
	unsigned int d_in_len,
	unsigned int max_elems_per_block)
{
	// need shared memory array for:
	// - block's share of the input data (local sort will be put here too)
	// - mask outputs
	// - scanned mask outputs
	// - merged scaned mask outputs ("local prefix sum")
	// - local sums of scanned mask outputs
	// - scanned local sums of scanned mask outputs

	// for all radix combinations:
	//  build mask output for current radix combination
	//  scan mask ouput
	//  store needed value from current prefix sum array to merged prefix sum array
	//  store total sum of mask output (obtained from scan) to global block sum array
	// calculate local sorted address from local prefix sum and scanned mask output's total sums
	// shuffle input block according to calculated local sorted addresses
	// shuffle local prefix sums according to calculated local sorted addresses
	// copy locally sorted array back to global memory
	// copy local prefix sum array back to global memory
}

__global__ void gpu_glbl_shuffle()
{
	// get d = digit
	// get n = blockIdx
	// get m = local prefix sum array value
	// calculate global position P_d[n] + m
	// copy input element to final position in d_out
}

// An attempt at the gpu radix sort variant described in this paper:
// https://vgc.poly.edu/~csilva/papers/cgf.pdf
void radix_sort_4way()
{
	// for every 2 bits from LSB to MSB:
	//  block-wise radix sort (write blocks back to global memory)

	//  scan global block sum array

	//  scatter/shuffle block-wise sorted array to final positions

	//  copy d_out to d_in in prep for next pass
}

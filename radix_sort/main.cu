#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <algorithm>
#include <iostream>

#include "sort.h"
#include "utils.h"

void cpu_sort(unsigned int* h_out, unsigned int* h_in, size_t len)
{
	for (int i = 0; i < len; ++i)
	{
		h_out[i] = h_in[i];
	}

	std::sort(h_out, h_out + len);
}

int main()
{
	// Set up clock for timing comparisons
	srand(time(NULL));
	std::clock_t start;
	double duration;

	for (int i = 0; i < 28; ++i)
	{
		unsigned int num_elems = (1 << i) - 1;
		//unsigned int num_elems = 8192;
		std::cout << "h_in size: " << num_elems << std::endl;

		unsigned int* h_in = new unsigned int[num_elems];
		unsigned int* h_out_cpu = new unsigned int[num_elems];
		unsigned int* h_out_gpu = new unsigned int[num_elems];

		for (int i = 0; i < num_elems; i++)
		{
			h_in[i] = (num_elems - 1) - i;
			//std::cout << h_in[i] << " ";
		}
		start = std::clock();
		cpu_sort(h_out_cpu, h_in, num_elems);
		duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
		std::cout << "CPU time: " << duration << std::endl;

		unsigned int* d_in;
		unsigned int* d_preds;
		unsigned int* d_scanned_preds;
		unsigned int* d_out;
		checkCudaErrors(hipMalloc(&d_in, sizeof(unsigned int) * num_elems));
		checkCudaErrors(hipMalloc(&d_preds, sizeof(unsigned int) * num_elems));
		checkCudaErrors(hipMalloc(&d_scanned_preds, sizeof(unsigned int) * num_elems));
		checkCudaErrors(hipMalloc(&d_out, sizeof(unsigned int) * num_elems));
		checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(unsigned int) * num_elems, hipMemcpyHostToDevice));
		start = std::clock();
		radix_sort(d_out, d_in, num_elems);
		duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
		std::cout << "GPU time: " << duration << std::endl;
		checkCudaErrors(hipMemcpy(h_out_gpu, d_out, sizeof(unsigned int) * num_elems, hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_out));
		checkCudaErrors(hipFree(d_scanned_preds));
		checkCudaErrors(hipFree(d_preds));
		checkCudaErrors(hipFree(d_in));

		// Check for any mismatches between outputs of CPU and GPU
		bool match = true;
		int index_diff = 0;
		for (int i = 0; i < num_elems; ++i)
		{
			if (h_out_cpu[i] != h_out_gpu[i])
			{
				match = false;
				index_diff = i;
				break;
			}
		}
		std::cout << "Match: " << match << std::endl;

		// Detail the mismatch if any
		if (!match)
		{
			std::cout << "Difference in index: " << index_diff << std::endl;
			std::cout << "CPU: " << h_out_cpu[index_diff] << std::endl;
			std::cout << "GPU Radix Sort: " << h_out_gpu[index_diff] << std::endl;
			int window_sz = 10;

			std::cout << "Contents: " << std::endl;
			std::cout << "CPU: ";
			for (int i = -(window_sz / 2); i < (window_sz / 2); ++i)
			{
				std::cout << h_out_cpu[index_diff + i] << ", ";
			}
			std::cout << std::endl;
			std::cout << "GPU Radix Sort: ";
			for (int i = -(window_sz / 2); i < (window_sz / 2); ++i)
			{
				std::cout << h_out_gpu[index_diff + i] << ", ";
			}
			std::cout << std::endl;
		}

		delete[] h_out_gpu;
		delete[] h_out_cpu;
		delete[] h_in;

		std::cout << std::endl;
	}
}

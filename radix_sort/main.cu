#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <ctime>

#include "sort.h"
#include "utils.h"

#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

void cpu_sort(unsigned int* h_out, unsigned int* h_in, size_t len)
{
    for (int i = 0; i < len; ++i)
    {
        h_out[i] = h_in[i];
    }

    std::sort(h_out, h_out + len);
}

void test_cpu_vs_gpu(unsigned int* h_in, unsigned int num_elems)
{
    std::clock_t start;

    unsigned int* h_out_cpu = new unsigned int[num_elems];
    unsigned int* h_out_gpu = new unsigned int[num_elems];

    start = std::clock();
    cpu_sort(h_out_cpu, h_in, num_elems);
    double cpu_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
    std::cout << "CPU time: " << cpu_duration << " s" << std::endl;
    
    unsigned int* d_in;
    unsigned int* d_out;
    checkCudaErrors(hipMalloc(&d_in, sizeof(unsigned int) * num_elems));
    checkCudaErrors(hipMalloc(&d_out, sizeof(unsigned int) * num_elems));
    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(unsigned int) * num_elems, hipMemcpyHostToDevice));
    start = std::clock();
    radix_sort(d_out, d_in, num_elems);
    double gpu_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
    std::cout << "GPU time: " << gpu_duration << " s" << std::endl;
    checkCudaErrors(hipMemcpy(h_out_gpu, d_out, sizeof(unsigned int) * num_elems, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_in));

    hipEvent_t event_start, stop;
    hipEventCreate(&event_start);
    hipEventCreate(&stop);

    unsigned int* d_thrust_in;
    checkCudaErrors( hipMalloc(&d_thrust_in, sizeof(unsigned int) * num_elems) );
    checkCudaErrors( hipMemcpy(d_thrust_in, h_in, sizeof(unsigned int) * num_elems, hipMemcpyHostToDevice) );

    thrust::device_ptr<unsigned int> d_ptr_in(d_thrust_in);

    hipEventRecord(event_start);
    thrust::sort(d_ptr_in, d_ptr_in + num_elems);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, event_start, stop);
    milliseconds /= 1000.0;
    std::cout << "Thrust time is " << milliseconds << std::endl;
    std::cout << "Thrust Speedup cpu " << cpu_duration / milliseconds << "x" << std::endl; 
    std::cout << "Thrust Speedup 4-way " << gpu_duration / milliseconds << "x" << std::endl;

    // Calculate GPU / CPU speedup
    std::cout << "Speedup: " << cpu_duration / gpu_duration << "x" << std::endl;

    // Check for any mismatches between outputs of CPU and GPU
    bool match = true;
    int index_diff = 0;
    for (int i = 0; i < num_elems; ++i)
    {
        if (h_out_cpu[i] != h_out_gpu[i])
        {
            match = false;
            index_diff = i;
            break;
        }
    }
    std::cout << "Match: " << match << std::endl;
    
    // Detail the mismatch if any
    if (!match)
    {
        std::cout << "Difference in index: " << index_diff << std::endl;
        std::cout << "CPU: " << h_out_cpu[index_diff] << std::endl;
        std::cout << "GPU Radix Sort: " << h_out_gpu[index_diff] << std::endl;
        int window_sz = 10;
    
        std::cout << "Contents: " << std::endl;
        std::cout << "CPU: ";
        for (int i = -(window_sz / 2); i < (window_sz / 2); ++i)
        {
            std::cout << h_out_cpu[index_diff + i] << ", ";
        }
        std::cout << std::endl;
        std::cout << "GPU Radix Sort: ";
        for (int i = -(window_sz / 2); i < (window_sz / 2); ++i)
        {
            std::cout << h_out_gpu[index_diff + i] << ", ";
        }
        std::cout << std::endl;
    }
    
    delete[] h_out_gpu;
    delete[] h_out_cpu;
}

int main()
{
    // Set up clock for timing comparisons
    srand(1);

    for (int i = 20; i < 21; ++i)
    {
        unsigned int num_elems = (1 << i);
        //unsigned int num_elems = 8;
        std::cout << "h_in size: " << num_elems << std::endl;

        unsigned int* h_in = new unsigned int[num_elems];
        unsigned int* h_in_rand = new unsigned int[num_elems];

        for (int j = 0; j < num_elems; j++)
        {
            h_in[j] = (num_elems - 1) - j;
            h_in_rand[j] = rand() % num_elems;
            //std::cout << h_in[j] << " ";
        }
        //std::cout << std::endl;

        std::cout << "*** i: " << i << " ***" << std::endl;
        for (int j = 0; j < 5; ++j) {
            std::cout << "*****Descending order*****" << std::endl;
            test_cpu_vs_gpu(h_in, num_elems);
            std::cout << "*****Random order*****" << std::endl;
            test_cpu_vs_gpu(h_in_rand, num_elems);
            std::cout << std::endl;
        }

        delete[] h_in;
        delete[] h_in_rand;

        std::cout << std::endl;
    }
}
